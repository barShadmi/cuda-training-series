
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello(){

  printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

int main(){
  int blocks = 2, threads = 3;

  hello<<<blocks, threads>>>();
  hipDeviceSynchronize();
}

